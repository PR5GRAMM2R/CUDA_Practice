#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

__global__ void helloCUDA(void)
{
	printf("Hello CUDA (GPU)!\n");
}

int main()
{
	printf("Hello CUDA (CPU)!\n");

	helloCUDA << <1, 10 >> > ();
	hipDeviceSynchronize();

	return 0;
}